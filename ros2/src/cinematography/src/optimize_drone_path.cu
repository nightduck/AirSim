#include "hip/hip_runtime.h"
#include "optimize_drone_path.cuh"
#include <stdgpu/unordered_map.cuh>
#include <stdgpu/unordered_set.cuh>
#include <stdgpu/memory.h>

//error function for cpu called after kernel calls
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace stdgpu {
    template<>
    struct hash<Voxel>{
        inline STDGPU_HOST_DEVICE std::size_t
        operator()(const Voxel& v) const
        {
            Eigen::Matrix<double, 3, 1> p = v.position;
            return ((int)(p(0)*73856093) ^ (int)(p(1)*19349669) ^ (int)(p(2)*83492791));
            // using stdgpu::hash;
            // return(hash<double>()(k.x) ^ hash<double>()(k.y) ^ hash<double>()(k.z));
        }
    };
}

stdgpu::unordered_set<Voxel> set;

__global__ 
void init_set(stdgpu::unordered_set<Voxel> set, Voxel * voxels, int * voxels_size){
    int thread_index = (blockIdx.x*512 + threadIdx.x);
    if(thread_index >= *voxels_size){
        return;
    }
    Voxel v = voxels[thread_index];
    set.insert(v);
}

__device__
double floor_fun_cuda(const double & x, const double & scale){
    return floor(x*scale) / scale;
}

/*
* Given world point return center of volume given by volume_size
*/
__device__
Eigen::Matrix<double, 3, 1> get_volume_center_from_point_cuda(Eigen::Matrix<double, 3, 1> point, double volume_size){
    double scale = 1/volume_size;
    double half_volume_size = volume_size / 2;
    Eigen::Matrix<double, 3, 1> volume_center;
    volume_center(0) = floor_fun_cuda(point(0), scale) + half_volume_size;
    volume_center(1) = floor_fun_cuda(point(1), scale) + half_volume_size;
    volume_center(2) = floor_fun_cuda(point(2), scale) + half_volume_size;
    return volume_center;
}

/*
* Check if two Eigen::Matrix<double, 3, 1> are equal
*/
__device__
bool check_floating_point_vectors_equal_cuda(Eigen::Matrix<double, 3, 1> A, Eigen::Matrix<double, 3, 1> B, double epsilon){
    Eigen::Matrix<double, 3, 1> diff = A-B;
    //have to use an epsilon value due to floating point precision errors
    if((fabs(diff(0)) < epsilon) && (fabs(diff(1)) < epsilon) && (fabs(diff(2)) < epsilon))
    return true;

    return false;
}

/*
* Get voxels between point start and point end
* For more information on voxel traversal algo: http://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.42.3443&rep=rep1&type=pdf
*/
__device__
void get_voxels(const MultiDOF point_start, const MultiDOF point_end, Eigen::Matrix<double, 3, 1> * voxels, const double & volume_size, int & size){
    const double half_volume_size = volume_size / 2;
    const double epsilon = volume_size / 4;
    const double volume_size_plus_epsilon = volume_size + epsilon;
    const double volume_size_minus_epsilon = volume_size - epsilon;
    Eigen::Matrix<double, 3, 1> start(point_start.x, point_start.y, point_start.z);
    Eigen::Matrix<double, 3, 1> end(point_end.x, point_end.y, point_end.z);
    //   equation of line = u+tv
    Eigen::Matrix<double, 3, 1> u(point_start.x, point_start.y, point_start.z);
    Eigen::Matrix<double, 3, 1> v(point_end.x - point_start.x, point_end.y - point_start.y, point_end.z - point_start.z);
    double stepX = v(0) > 0 ? volume_size : -1 * volume_size;
    double stepY = v(1) > 0 ? volume_size : -1 * volume_size;
    double stepZ = v(2) > 0 ? volume_size : -1 * volume_size;
    Eigen::Matrix<double, 3, 1> start_voxel_center = get_volume_center_from_point_cuda(start, volume_size);
    double tMaxX = fabs(v(0) < 0 ? (start_voxel_center(0) - half_volume_size - u(0)) / v(0) : (start_voxel_center(0) + half_volume_size - u(0)) / v(0));
    double tMaxY = fabs(v(1) < 0 ? (start_voxel_center(1) - half_volume_size - u(1)) / v(1) : (start_voxel_center(1) + half_volume_size - u(1)) / v(1));
    double tMaxZ = fabs(v(2) < 0 ? (start_voxel_center(2) - half_volume_size - u(2)) / v(2) : (start_voxel_center(2) + half_volume_size - u(2)) / v(2));
    double tDeltaX = fabs(volume_size / v(0));
    double tDeltaY = fabs(volume_size / v(1));
    double tDeltaZ = fabs(volume_size / v(2));
    Eigen::Matrix<double, 3, 1> current_vol(start(0), start(1), start(2));
    Eigen::Matrix<double, 3, 1> current_vol_center = get_volume_center_from_point_cuda(current_vol, volume_size);
    Eigen::Matrix<double, 3, 1> end_voxel_center = get_volume_center_from_point_cuda(end, volume_size);

    while(!check_floating_point_vectors_equal_cuda(current_vol_center, end_voxel_center, epsilon)){
        //add traversed voxel key to list of voxel keys
        Eigen::Matrix<double, 3, 1> voxel = current_vol_center;
        voxels[size] = voxel;
        size++;
        
        if(tMaxX < tMaxY){
            if(tMaxX < tMaxZ)
            {
                current_vol(0) += stepX;
                tMaxX += tDeltaX;
            }
            else if(tMaxX > tMaxZ){
                current_vol(2) += stepZ;
                tMaxZ += tDeltaZ;
            }
            else{
                current_vol(0) += stepX;
                current_vol(2) += stepZ;
                tMaxX += tDeltaX;
                tMaxZ += tDeltaZ;
            }
        }
        else if(tMaxX > tMaxY){
            if(tMaxY < tMaxZ){
            current_vol(1) += stepY;
            tMaxY += tDeltaY;
            }
            else if(tMaxY > tMaxZ){
                current_vol(2) += stepZ;
                tMaxZ += tDeltaZ;
            }
            else{
                current_vol(1) += stepY;
                current_vol(2) += stepZ;
                tMaxY += tDeltaY;
                tMaxZ += tDeltaZ;
            }
        }
        else{
            if(tMaxZ < tMaxX){
                current_vol(2) += stepZ;
                tMaxZ += tDeltaZ;
            }
            else if(tMaxZ > tMaxX){
                current_vol(0) += stepX;
                current_vol(1) += stepY;
                tMaxX += tDeltaX;
                tMaxY += tDeltaY;
            }
            else{ 
                current_vol(0) += stepX;
                current_vol(1) += stepY;
                current_vol(2) += stepZ;
                tMaxX += tDeltaX;
                tMaxY += tDeltaY;
                tMaxZ += tDeltaZ;
            }
        } 
        //deals with floating point precision errors
        Eigen::Matrix<double, 3, 1> temp_current_vol_center = current_vol_center;
        current_vol_center = get_volume_center_from_point_cuda(current_vol, volume_size);
        Eigen::Matrix<double, 3, 1> diff;
        diff(0) = fabs(temp_current_vol_center(0) - current_vol_center(0));
        diff(1) = fabs(temp_current_vol_center(1) - current_vol_center(1));
        diff(2) = fabs(temp_current_vol_center(2) - current_vol_center(2));
        if((diff(0) < volume_size_minus_epsilon && diff(1) < volume_size_minus_epsilon && diff(2) < volume_size_minus_epsilon) 
        || (diff(0) > volume_size_plus_epsilon || diff(1) > volume_size_plus_epsilon || diff(2) > volume_size_plus_epsilon))
        {
            return;
        }
    }      

    //add traversed voxel key to list of voxel keys
    Eigen::Matrix<double, 3, 1> voxel = current_vol_center;
    voxels[size] = voxel;
    size++;
}

/*
* Cost of sdf value
*/
__device__
double get_cost(const double & sdf, const double & truncation_distance){
    if(fabs(sdf) >= truncation_distance){
        return 0;
    }
    else if(sdf > 0){
        return pow((sdf - truncation_distance), 2) / (2* truncation_distance);
    }else{
        return sdf * -1 + .5 * truncation_distance;
    }

}

__device__
inline double get_voxel_cost(const Eigen::Matrix<double, 3, 1> & voxel_pos, const double & volume_size, stdgpu::unordered_set<Voxel> set, double * truncation_distance){

    Voxel voxel(0, voxel_pos(0), voxel_pos(1), voxel_pos(2));
    const Voxel * result = set.find(voxel);
    if(result != set.end()){
        return get_cost(result->sdf, * truncation_distance);
    }
    return 0; //voxel does not exist so it is in free space(or inside an object) and return 0 cost
}

/*
* Compute cost gradient for a voxel. Check cost values of voxel and voxels around
*/
__device__
Eigen::Matrix<double, 3, 1> get_voxel_cost_gradient(const Eigen::Matrix<double, 3, 1> & voxel_pos, const double & volume_size, stdgpu::unordered_set<Voxel> set, double * truncation_distance){
    double cost = get_voxel_cost(voxel_pos, volume_size, set, truncation_distance);
    Eigen::Matrix<double, 3, 1> gradient_val;

    Eigen::Matrix<double, 3, 1> xNext(voxel_pos(0) + volume_size, voxel_pos(1), voxel_pos(2));
    Eigen::Matrix<double, 3, 1> xPrev(voxel_pos(0) - volume_size, voxel_pos(1), voxel_pos(2));
    double xDiffNext = get_voxel_cost(xNext, volume_size, set, truncation_distance) - cost;
    double xDiffPrev = cost - get_voxel_cost(xPrev, volume_size, set, truncation_distance);
    gradient_val(0) = (xDiffNext + xDiffPrev) / 2;

    Eigen::Matrix<double, 3, 1> yNext(voxel_pos(0), voxel_pos(1) + volume_size, voxel_pos(2));
    Eigen::Matrix<double, 3, 1> yPrev(voxel_pos(0), voxel_pos(1) - volume_size, voxel_pos(2));
    double yDiffNext = get_voxel_cost(yNext, volume_size, set, truncation_distance) - cost;
    double yDiffPrev = cost - get_voxel_cost(yPrev, volume_size, set, truncation_distance);
    gradient_val(1) = (yDiffNext + yDiffPrev) / 2;

    Eigen::Matrix<double, 3, 1> zNext(voxel_pos(0), voxel_pos(1), voxel_pos(2) + volume_size);
    Eigen::Matrix<double, 3, 1> zPrev(voxel_pos(0), voxel_pos(1), voxel_pos(2) - volume_size);
    double zDiffNext = get_voxel_cost(zNext, volume_size, set, truncation_distance) - cost;
    double zDiffPrev = cost - get_voxel_cost(zPrev, volume_size, set, truncation_distance);
    gradient_val(2) = (zDiffNext + zDiffPrev) / 2;

    return gradient_val;
}

__global__
void process_obstacle_avoidance_voxels(Eigen::Matrix<double, 3, 1> * voxels, int * size_p, stdgpu::unordered_set<Voxel> set, double * velocity_mag, 
    Eigen::Matrix<double, 3, 3> * identity_minus_p_hat_multiplied, double * obs_grad, int * point_index, double * truncation_distance, double * voxel_size){
    int thread_index = (blockIdx.x*256 + threadIdx.x);
    if(thread_index >=*size_p){
        return;
    }
    Eigen::Matrix<double, 3, 1> cost_function_gradient = get_voxel_cost_gradient(voxels[thread_index], * voxel_size, set, truncation_distance); //TODO: volume_size
    Eigen::Matrix<double, 3, 1> gradient_multiplied_result = (*identity_minus_p_hat_multiplied) * cost_function_gradient;       
    Eigen::Matrix<double, 3, 1> grad_j_obs = gradient_multiplied_result * (*velocity_mag);
    
    if(isnan(grad_j_obs(0)) || isnan(grad_j_obs(1)) || isnan(grad_j_obs(2))){
        grad_j_obs(0) = 0;
        grad_j_obs(1) = 0;
        grad_j_obs(2) = 0;
    }

    int obs_grad_index = (* point_index) * 3;

    atomicAdd(&(obs_grad[obs_grad_index]), grad_j_obs(0));
    atomicAdd(&(obs_grad[obs_grad_index + 1]), grad_j_obs(1));
    atomicAdd(&(obs_grad[obs_grad_index + 2]), grad_j_obs(2));

}

__global__
void obstacle_avoidance_gradient(MultiDOF * drone_traj, double * obs_grad, int * n, stdgpu::unordered_set<Voxel> set, double * truncation_distance, double * voxel_size){
    int thread_index = (blockIdx.x*128 + threadIdx.x);
    if(thread_index >= *n-1){
        return;
    }

    MultiDOF point_start = drone_traj[thread_index];
    MultiDOF point_end = drone_traj[thread_index+1];

    Eigen::Matrix<double, 3, 1> * voxels = new Eigen::Matrix<double, 3, 1>[500]; //figure out how to bound this
    int size = 0;
    get_voxels(point_start, point_end, voxels, * voxel_size, size);

    double velocity_mag = sqrt(pow(point_end.vx, 2) + pow(point_end.vy ,2) + pow(point_end.vz , 2));
    double * velocity_mag_p = new double(velocity_mag);
    Eigen::Matrix<double, 3, 1> p_hat(point_end.vx/velocity_mag, point_end.vy/velocity_mag, point_end.vz/velocity_mag);
    if(isnan(p_hat(0)) || isnan(p_hat(1)) || isnan(p_hat(2))){
      p_hat(0) = 0;
      p_hat(1) = 0;
      p_hat(2) = 0;
    }

    Eigen::Matrix<double, 3, 3> p_hat_multiplied = p_hat * p_hat.transpose();

    Eigen::Matrix<double, 3, 3> * identity_minus_p_hat_multiplied_p = new Eigen::Matrix3d();
    * identity_minus_p_hat_multiplied_p = Eigen::Matrix3d::Identity(3,3) - p_hat_multiplied;

    int threads_per_block = 256;
    int num_blocks = size / threads_per_block + 1;
    int * size_p = new int(size);
    int * thread_index_p = new int(thread_index);
    process_obstacle_avoidance_voxels<<<num_blocks, threads_per_block>>>(voxels, size_p, set, velocity_mag_p, 
        identity_minus_p_hat_multiplied_p, obs_grad, thread_index_p, truncation_distance, voxel_size);
    hipDeviceSynchronize();

    int obs_grad_index = thread_index * 3;

    obs_grad[obs_grad_index]/=size;
    obs_grad[obs_grad_index+1]/=size;
    obs_grad[obs_grad_index+2]/=size;

    free(voxels);
    free(velocity_mag_p);
    free(size_p);
    free(thread_index_p);
    free(identity_minus_p_hat_multiplied_p);
}

__global__
void process_occlusion_avoidance_voxels(Eigen::Matrix<double, 3, 1> * voxels, int * size_p, stdgpu::unordered_set<Voxel> set, 
     double * occ_grad, int * point_index, MultiDOF * point_start_p, MultiDOF * point_end_p, double * truncation_distance, double * voxel_size){
    int thread_index = (blockIdx.x*256 + threadIdx.x);
    if(thread_index >=*size_p){
        return;
    }

    MultiDOF point_start = * point_start_p;
    MultiDOF point_end = * point_end_p;

    Eigen::Matrix<double, 3, 1> actor_point_velocity(point_end.vx, point_end.vy, point_end.vz);

    Eigen::Matrix<double, 3, 1> drone_point_velocity(point_start.vx, point_start.vy, point_start.vz);
    double drone_point_velocity_mag = drone_point_velocity.norm();
    Eigen::Matrix<double, 3, 1> normalized_drone_point_velocity = drone_point_velocity/drone_point_velocity_mag; 
    if(isnan(normalized_drone_point_velocity(0)) || isnan(normalized_drone_point_velocity(1)) || isnan(normalized_drone_point_velocity(2))){
        normalized_drone_point_velocity(0) = 0;
        normalized_drone_point_velocity(1) = 0;
        normalized_drone_point_velocity(2) = 0;
    }
    
    Eigen::Matrix<double, 1, 3> normalized_drone_point_velocity_transpose = normalized_drone_point_velocity.transpose();

    Eigen::Matrix<double, 3, 1> L(point_end.x - point_start.x, point_end.y - point_start.y, point_end.z - point_start.z);
    double L_mag = L.norm();
    Eigen::Matrix<double, 3, 1> normalized_L = L/L_mag;
    Eigen::Matrix<double, 1, 3> normalized_L_transpose = normalized_L.transpose();
    Eigen::Matrix<double, 3, 1> L_velocity = actor_point_velocity - drone_point_velocity;

    //used for determining the value of τ
    double increment = 1.0/(*size_p-1); 

    Eigen::Matrix<double, 3, 1> cost_function_gradient = get_voxel_cost_gradient(voxels[thread_index], * voxel_size, set, truncation_distance);
    // Eigen::Matrix<double, 3, 1> cost_function_gradient(1,1,1);
    Eigen::Matrix<double, 3, 1> inner_first_term = actor_point_velocity/drone_point_velocity_mag - normalized_drone_point_velocity;
    double temp1 = inner_first_term(0);
    double temp2 = inner_first_term(1);
    double temp3 = inner_first_term(2);
    inner_first_term*=thread_index*increment;
    inner_first_term +=normalized_drone_point_velocity;
    Eigen::Matrix<double, 3, 3> inner_first_term_matrix = inner_first_term * normalized_drone_point_velocity_transpose;
    inner_first_term_matrix = Eigen::Matrix3d::Identity(3,3) - inner_first_term_matrix;
    Eigen::Matrix<double, 1, 3> first_term = cost_function_gradient.transpose() * L_mag * drone_point_velocity_mag * inner_first_term_matrix;

    Eigen::Matrix<double, 1, 3> inner_second_term = normalized_L_transpose + normalized_L_transpose * L_velocity * normalized_drone_point_velocity_transpose;
    Eigen::Matrix<double, 1, 3> second_term = get_voxel_cost(voxels[thread_index], * voxel_size, set, truncation_distance) * drone_point_velocity_mag * inner_second_term;
    // Eigen::Matrix<double, 1, 3> second_term(1,1,1);
    Eigen::Matrix<double, 1, 3> grad_j_occ = first_term - second_term;

    if(isnan(grad_j_occ(0)) || isnan(grad_j_occ(1)) || isnan(grad_j_occ(2))){
        grad_j_occ(0) = 0;
        grad_j_occ(1) = 0;
        grad_j_occ(2) = 0;
    }

    int occ_grad_index = (* point_index) * 3;

    atomicAdd(&(occ_grad[occ_grad_index]), grad_j_occ(0));
    atomicAdd(&(occ_grad[occ_grad_index + 1]), grad_j_occ(1));
    atomicAdd(&(occ_grad[occ_grad_index + 2]), grad_j_occ(2));

}

__global__ 
void occlusion_avoidance_gradient(MultiDOF * drone_traj, MultiDOF * actor_traj, double * occ_grad, int * n, stdgpu::unordered_set<Voxel> set, 
double * truncation_distance, double * voxel_size){
    int thread_index = (blockIdx.x*128 + threadIdx.x);
    if(thread_index >= *n-1){
        return;
    }
    
    MultiDOF * point_start_p = new MultiDOF(drone_traj[thread_index+1]); 
    // * point_start_p = drone_traj[thread_index+1];

    MultiDOF * point_end_p = new MultiDOF(actor_traj[thread_index+1]); 
    // * point_end_p = actor_traj[thread_index+1];

    Eigen::Matrix<double, 3, 1> * voxels = new Eigen::Matrix<double, 3, 1>[500]; //figure out how to bound this
    int size = 0;
    get_voxels(* point_start_p, * point_end_p, voxels, * voxel_size, size);

    int threads_per_block = 256;
    int num_blocks = size / threads_per_block + 1;
    int * size_p = new int(size);
    int * thread_index_p = new int(thread_index);
    process_occlusion_avoidance_voxels<<<num_blocks, threads_per_block>>>(voxels, size_p, set, occ_grad, 
        thread_index_p, point_start_p, point_end_p, truncation_distance, voxel_size);
    hipDeviceSynchronize();

    int occ_grad_index = thread_index * 3;

    occ_grad[occ_grad_index]/=size;
    occ_grad[occ_grad_index+1]/=size;
    occ_grad[occ_grad_index+2]/=size;

    free(voxels);
    free(size_p);
    free(thread_index_p);
    free(point_start_p);
    free(point_end_p);
}

void allocate_set(){
    set = stdgpu::unordered_set<Voxel>::createDeviceObject(5000000);
}

void init_set_cuda(std::vector<Voxel> & voxels){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    set.clear();

    int * voxels_size_h = new int(voxels.size());

    if(*voxels_size_h == 0){
        return;
    }

    // printf("voxels size: %d\n", * voxels_size_h);
    int * voxels_size_d;
    hipMalloc(&voxels_size_d, sizeof(*voxels_size_h));
    hipMemcpy(voxels_size_d, voxels_size_h, sizeof(*voxels_size_h), hipMemcpyHostToDevice);

    Voxel * voxels_h = &voxels[0];
    Voxel * voxels_d;
    hipMalloc(&voxels_d, sizeof(*voxels_h) * (*voxels_size_h));
    hipMemcpy(voxels_d, voxels_h, sizeof(*voxels_h) * (*voxels_size_h), hipMemcpyHostToDevice);

    int num_threads = 512;
    int num_blocks = *voxels_size_h / num_threads + 1;
    // printf("max size: %d\n", set.max_size());
    init_set<<<num_blocks,num_threads>>>(set, voxels_d, voxels_size_d);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    // printf("valid? : %d\n", set.valid());
    printf("set size: %d\n", set.size());
    printf("set load factor: %f\n", set.load_factor());
    // printf("set max load factor: %f\n", set.max_load_factor());
    printf("num buckets: %d\n", set.bucket_count());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("set generation: %f\n", milliseconds);

}

Eigen::Matrix<double, Eigen::Dynamic, 3> obstacle_avoidance_gradient_cuda(std::vector<MultiDOF>  & drone_traj, double & truncation_distance, double & voxel_size){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int * n_h = new int(drone_traj.size());
    Eigen::Matrix<double, Eigen::Dynamic, 3> obs_grad((*n_h)-1,3);

    if(set.size() == 0){
        for(int i=0;i<(*n_h)-1; ++i){
            obs_grad(i,0) = 0;
            obs_grad(i,1) = 0;
            obs_grad(i,2) = 0;
        }
        return obs_grad;
    }
    
    int * n_d;
    hipMalloc(&n_d, sizeof(*n_h));
    hipMemcpy(n_d, n_h, sizeof(*n_h), hipMemcpyHostToDevice);

    MultiDOF * drone_traj_h = &drone_traj[0];
    MultiDOF * drone_traj_d;

    hipMalloc(&drone_traj_d, sizeof(*drone_traj_h)*(*n_h));
    hipMemcpy(drone_traj_d, drone_traj_h, sizeof(*drone_traj_h)*(*n_h), hipMemcpyHostToDevice);

    double * truncation_distance_h = new double(truncation_distance);
    double * truncation_distance_d;
    hipMalloc(&truncation_distance_d, sizeof(*truncation_distance_h));
    hipMemcpy(truncation_distance_d, truncation_distance_h, sizeof(*truncation_distance_h), hipMemcpyHostToDevice);

    double * voxel_size_h = new double(voxel_size);
    double * voxel_size_d;
    hipMalloc(&voxel_size_d, sizeof(*voxel_size_h));
    hipMemcpy(voxel_size_d, voxel_size_h, sizeof(*voxel_size_h), hipMemcpyHostToDevice);

    int obs_grad_size = ((*n_h)-1) * 3;
    double obs_grad_h[obs_grad_size];
    double * obs_grad_d;
    hipMalloc(&obs_grad_d, sizeof(*obs_grad_h) * obs_grad_size);
    hipMemset(obs_grad_d, 0, sizeof(*obs_grad_h) * obs_grad_size);

    int num_threads = 128;
    int num_blocks = *n_h / num_threads + 1;

    obstacle_avoidance_gradient<<<num_blocks, num_threads>>>(drone_traj_d, obs_grad_d, n_d, set, truncation_distance_d, voxel_size_d);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    hipMemcpy(obs_grad_h, obs_grad_d, sizeof(*obs_grad_h) * obs_grad_size, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    for(int i=0;i<(*n_h)-1; ++i){
        int obs_grad_h_index = 3*i;
        obs_grad(i,0) = obs_grad_h[obs_grad_h_index];
        obs_grad(i,1) = obs_grad_h[obs_grad_h_index + 1];
        obs_grad(i,2) = obs_grad_h[obs_grad_h_index + 2];
    }

    free(n_h);
    free(truncation_distance_h);
    free(voxel_size_h);
    hipFree(n_d);
    hipFree(drone_traj_d);
    hipFree(obs_grad_d);
    hipFree(truncation_distance_d);
    hipFree(voxel_size_d);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("obstacle avoidance gradient duration: %f\n", milliseconds);

    return obs_grad;
}

Eigen::Matrix<double, Eigen::Dynamic, 3> occlusion_avoidance_gradient_cuda(std::vector<MultiDOF>  & drone_traj, 
    std::vector<MultiDOF> & actor_traj, double & truncation_distance, double & voxel_size){
    
    int * n_h = new int(drone_traj.size());
    Eigen::Matrix<double, Eigen::Dynamic, 3> occ_grad((*n_h)-1,3);
    
    if(set.size() == 0){
        for(int i=0;i<(*n_h)-1; ++i){
            occ_grad(i,0) = 0;
            occ_grad(i,1) = 0;
            occ_grad(i,2) = 0;
        }
        return occ_grad;
    }

    int * n_d;
    hipMalloc(&n_d, sizeof(*n_h));
    hipMemcpy(n_d, n_h, sizeof(*n_h), hipMemcpyHostToDevice);

    MultiDOF * drone_traj_h = &drone_traj[0];
    MultiDOF * actor_traj_h = &actor_traj[0];
    MultiDOF * drone_traj_d;
    MultiDOF * actor_traj_d;

    hipMalloc(&drone_traj_d, sizeof(*drone_traj_h)*(*n_h));
    hipMemcpy(drone_traj_d, drone_traj_h, sizeof(*drone_traj_h)*(*n_h), hipMemcpyHostToDevice);
    hipMalloc(&actor_traj_d, sizeof(*actor_traj_h)*(*n_h));
    hipMemcpy(actor_traj_d, actor_traj_h, sizeof(*actor_traj_h)*(*n_h), hipMemcpyHostToDevice);

    double * truncation_distance_h = new double(truncation_distance);
    double * truncation_distance_d;
    hipMalloc(&truncation_distance_d, sizeof(*truncation_distance_h));
    hipMemcpy(truncation_distance_d, truncation_distance_h, sizeof(*truncation_distance_h), hipMemcpyHostToDevice);

    double * voxel_size_h = new double(voxel_size);
    double * voxel_size_d;
    hipMalloc(&voxel_size_d, sizeof(*voxel_size_h));
    hipMemcpy(voxel_size_d, voxel_size_h, sizeof(*voxel_size_h), hipMemcpyHostToDevice);

    int occ_grad_size = ((*n_h)-1) * 3;
    double occ_grad_h[occ_grad_size];
    double * occ_grad_d;
    hipMalloc(&occ_grad_d, sizeof(*occ_grad_h) * occ_grad_size);
    hipMemset(occ_grad_d, 0, sizeof(*occ_grad_h) * occ_grad_size);

    int num_threads = 128;
    int num_blocks = *n_h / num_threads + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    occlusion_avoidance_gradient<<<num_blocks, num_threads>>>(drone_traj_d, actor_traj_d, occ_grad_d, n_d, set, truncation_distance_d, voxel_size_d);
    gpuErrchk(hipPeekAtLastError());
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    // printf("occlusion avoidance gradient duration: %f\n", milliseconds);

    hipMemcpy(occ_grad_h, occ_grad_d, sizeof(*occ_grad_h) * occ_grad_size, hipMemcpyDeviceToHost);
    gpuErrchk(hipPeekAtLastError());

    for(int i=0;i<(*n_h)-1; ++i){
        int occ_grad_h_index = 3*i;
        occ_grad(i,0) = occ_grad_h[occ_grad_h_index];
        occ_grad(i,1) = occ_grad_h[occ_grad_h_index + 1];
        occ_grad(i,2) = occ_grad_h[occ_grad_h_index + 2];
    }

    free(n_h);
    free(truncation_distance_h);
    free(voxel_size_h);
    hipFree(n_d);
    hipFree(drone_traj_d);
    hipFree(actor_traj_d);
    hipFree(occ_grad_d);
    hipFree(truncation_distance_d);
    hipFree(voxel_size_d);

    return occ_grad;
}